#include <SpSolver.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <mkl.h>

//TODO: cuda error handling
int cuSolverSpSolver(SpMat &A, double* b, double* x) {
  hipsolverSpHandle_t handle; hipsolverSpCreate(&handle);
  
  hipsparseMatDescr_t descrA; hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);  

  int reorder=0;
  double tol=0;
  int singularity=0;

  hipsolverStatus_t t = cusolverSpDcsrlsvluHost(handle,A.n,A.nnz,descrA,A.csrVal.data(),A.csrRowPtr.data(),A.csrColInd.data(),b,tol,reorder,x,&singularity);
  printf("singularity: %d\n", singularity);

  return t;
}


//Todo MKL Error handling
int mklPardiso(SpMat &A, double* b, double* x) {
  //pardiso(_MKL_DSS_HANDLE_t pt, const MKL_INT *maxfct, const MKL_INT *mnum, const MKL_INT *mtype, const MKL_INT *phase, const MKL_INT *n, const void *a, const MKL_INT *ia, const MKL_INT *ja, MKL_INT *perm, const MKL_INT *nrhs, MKL_INT *iparm, const MKL_INT *msglvl, void *b, void *x, MKL_INT *error);
  void *pt[64];
  MKL_INT maxfct=1, mnum=1, mtype=11, n = A.n, perm=1, nrhs=1, msglvl=0, error=0;
  MKL_INT iparm[64];
  for(int i=0; i<64; i++) pt[i]=0, iparm[i]=0;
  pardisoinit(pt, &mtype, iparm);
  iparm[34]=1;

  std::vector<MKL_INT> RowPtr(A.csrRowPtr.size());
  std::vector<MKL_INT> ColInd(A.csrColInd.size());
  for(int i=0; i<RowPtr.size(); i++) RowPtr[i] = A.csrRowPtr[i]; //Todo int vector->long long vector casting?
  for(int i=0; i<ColInd.size(); i++) ColInd[i] = A.csrColInd[i];
  
  MKL_INT phase=13;
  pardiso(pt, &maxfct, &mnum, &mtype, &phase, &n, A.csrVal.data(), RowPtr.data(), ColInd.data(), &perm, &nrhs, iparm, &msglvl, b, x, &error);
  return (int)error;
}